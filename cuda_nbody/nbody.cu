#include <iostream>
#include <fstream>
#include <math.h>
#include <hip/hip_runtime.h>
#include <random>

using namespace std;

// Returns the square of the distance between two particles
__device__ float distance2(const float3 p1, const float3 p2)
{
  float3 r;
  r.x = p2.x - p1.x;
  r.y = p2.y - p1.y;
  r.z = p2.z - p1.z;

  return r.x*r.x + r.y*r.y + r.z*r.z;
}

// Calculate the force between two particles. It's repulsive when too close and
// attractive when far
__device__ float3 force(const float3 from, const float3 on, float coefficient)
{
  float d2 = distance2(from, on);
  float3 f;

  if(d2 < 100)
  {
    // Repel
    f.x = -coefficient * (from.x - on.x) / d2;
    f.y = -coefficient * (from.y - on.y) / d2;
    f.z = -coefficient * (from.z - on.z) / d2;
  }
  else if(d2 > 101)
  {
    // Attract
    f.x = coefficient * (from.x - on.x) / d2;
    f.y = coefficient * (from.y - on.y) / d2;
    f.z = coefficient * (from.z - on.z) / d2;
  }
  else
  {
    // No force
    f.x = 0;
    f.y = 0;
    f.z = 0;
  }

  return f;
}

// Perform an iteration of the simulation
__global__ void iterate(const int num_particles, const int num_iterations, const float coefficient, const float damping, float3 *particles, float3 *velocities)
{
  // Use shared memory. There is only one "extern".
  // You need to divide it up yourself
  extern __shared__ float3 sharedmem[];
  float3 *localparticles = sharedmem;
  float3 *localvelocities = &sharedmem[blockDim.x];
  float3 *localforces = &sharedmem[2*blockDim.x];

  // Get the index the shared data
  int sid = threadIdx.x;

  // Get the index the global data
  int gid = blockIdx.x * blockDim.x + threadIdx.x;

  // Copy the global memory to the shared memory of the block
  if(gid < num_particles)
  {
    localparticles[sid].x = particles[gid].x;
    localparticles[sid].y = particles[gid].y;
    localparticles[sid].z = particles[gid].z;
    localvelocities[sid].x = velocities[gid].x;
    localvelocities[sid].y = velocities[gid].y;
    localvelocities[sid].z = velocities[gid].z;
  }

  // Wait until all threads have copied the data
  __syncthreads();

  // Do some physics
  
  if(gid < num_particles)
  {
    for(int j = 0; j < num_iterations; j++)
    {
      for(int i = 0; i < blockDim.x; i++)
      {
        if(i == sid)
          continue;

        localforces[sid] = force(localparticles[i], localparticles[sid], coefficient);
        
      }
      __syncthreads();
      localvelocities[sid].x = (1 - damping) * (localvelocities[sid].x + localforces[sid].x);
      localvelocities[sid].y = (1 - damping) * (localvelocities[sid].y + localforces[sid].y);
      localvelocities[sid].z = (1 - damping) * (localvelocities[sid].z + localforces[sid].z);
      localparticles[sid].x += localvelocities[sid].x;
      localparticles[sid].y += localvelocities[sid].y;
      localparticles[sid].z += localvelocities[sid].z;
    }
  }

  // Wait until all threads have finished the simulation
  __syncthreads();

  // Copy the shared memory back to the block
  if(gid < num_particles)
  {
    particles[gid].x = localparticles[sid].x;
    particles[gid].y = localparticles[sid].y;
    particles[gid].z = localparticles[sid].z;
    velocities[gid].x = localvelocities[sid].x;
    velocities[gid].y = localvelocities[sid].y;
    velocities[gid].z = localvelocities[sid].z;
  }
}

void writeParticles(float3 *particles, float3 *velocities, int num_particles, int iteration_number)
{
  ofstream out("particles_" + to_string(iteration_number) + ".csv");
  out << "x,y,z,vx,vy,vz\n";
  for(int i = 0; i < num_particles; i++)
  {
    out << particles[i].x << "," << particles[i].y << "," << particles[i].z << ",";
    out << velocities[i].x << "," << velocities[i].y << "," << velocities[i].z << "\n";
  }
  out.close();
}

int main(void)
{
  const int num_particles = 512;
  const float coefficient = 0.001;
  const float damping = 0.005;
  const int num_iterations = 4000;
  const int iterations_per_write = 20;
  float3 *particles;
  float3 *velocities;
  
  // Allocate Unified Memory
  // This is automatically synced between host and GPU
  hipError_t err = hipMallocManaged(&particles, num_particles*sizeof(float3));
  if(err != hipSuccess) {
    cerr << "Failed to allocated memory on GPU: " << hipGetErrorString(err) << endl;
    return 1;
  }
  err = hipMallocManaged(&velocities, num_particles*sizeof(float3));
  if(err != hipSuccess) {
    cerr << "Failed to allocated memory on GPU: " << hipGetErrorString(err) << endl;
    return 1;
  }

  std::default_random_engine generator;
  std::uniform_real_distribution<double> dist(-10, 10);

  // Initialize the arrays
  for(int i = 0; i < num_particles; i++) {
    particles[i].x = dist(generator);
    particles[i].y = dist(generator);
    particles[i].z = dist(generator);
  }
  for(int i = 0; i < num_particles; i++) {
    velocities[i].x = 0.;
    velocities[i].y = 0.;
    velocities[i].z = 0.;
  }
  
  hipDeviceSynchronize();
  cout << "Data written" << endl;

  // Run kernel on the GPU
  int blockSize = 512;
  int numBlocks = (num_particles + blockSize - 1) / blockSize;
  int sharedMemSize = 3*(num_particles / numBlocks) * sizeof(float3);
  cout << "Requesting " << sharedMemSize << " bytes" << endl;
  cout << "blockSize: " << blockSize << endl;
  cout << "numBlocks: " << numBlocks << endl;
  
  writeParticles(particles, velocities, num_particles, 0);
  for(int i = 1; i <= num_iterations / iterations_per_write; i++)
  {
    cout << "Iteration " << (i * iterations_per_write) << endl;
    iterate<<<numBlocks, blockSize, sharedMemSize>>>(num_particles, iterations_per_write, coefficient, damping, particles, velocities);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    writeParticles(particles, velocities, num_particles, i);
  }
  cout << "Done" << endl;

  // Free memory
  hipFree(particles);
  hipFree(velocities);

  return 0;
} 

